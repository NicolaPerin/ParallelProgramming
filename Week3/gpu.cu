#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

void alloc(const int* row_counts, const int n_loc, const int rank, const int N, const double *A,
                        double **d_A, double **d_B_col, double **d_C) {
    // Allocate memory on the device
    hipMalloc( (void **)d_A, row_counts[rank] * N * sizeof(double) );
    hipError_t err = hipMalloc((void **)d_B_col, N * (n_loc + 1) * sizeof(double));
    if (err != hipSuccess) printf("Error allocating memory on the device: %s\n", hipGetErrorString(err));
    hipMalloc( (void **)d_C, row_counts[rank] * N * sizeof(double) );
    // copy A to the device (C is already allocated on the gpu)
    hipError_t err1 = hipMemcpy(*d_A, A, row_counts[rank] * N * sizeof(double), hipMemcpyHostToDevice);
    if (err1 != hipSuccess) { printf("Error on copying A to d_A: %s\n", hipGetErrorString(err1)); }
}

void gpu_computation(const int rank, const int p, const int n_loc, const int N, const int *row_counts,
                     const int *displ_B, const double *A, const double *B_col, double *C, double *d_A,
                     double* d_B_col, double *d_C, hipblasHandle_t handle) {

    if (B_col == NULL) printf("B_col pointer is null\n");
    if (d_B_col == NULL) printf("d_B_col pointer is null\n");
    // copy B_loc to the device (A is already allocated on the gpu)
    hipError_t err2 = hipMemcpy(d_B_col, B_col, N * (n_loc + 1) * sizeof(double), hipMemcpyHostToDevice);
    if (err2 != hipSuccess) { printf("Error on copying B_col to d_B_col: %s\n", hipGetErrorString(err2)); }
    // cublas multiplication
    const double alpha = 1.0, beta = 0.0;
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, row_counts[p], row_counts[rank], N, &alpha, d_B_col, row_counts[p], d_A, N, &beta, d_C + displ_B[p], N);
}
